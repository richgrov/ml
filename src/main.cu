
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello() {
    printf("Hello cuda\n");
}

int main() {
    hello<<<1, 1>>>();
    hipDeviceSynchronize();
    return 1;
}
